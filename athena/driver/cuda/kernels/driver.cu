EXPORT float* allocObj(float* arr, unsigned count){
    float *mem;
    const unsigned size = count*sizeof(float);
    hipMalloc(&mem, size);
    hipMemcpy(mem, arr, size, hipMemcpyHostToDevice);
    return mem;
};
EXPORT float* allocNum(unsigned count, float num){
    float *hostMem = (float*)malloc(sizeof(float)*count);
    for(int i=0; i<count; i++) hostMem[i] = num;
    float *deviceMem = allocObj(hostMem, count);
    free(hostMem);
    return deviceMem;
};
EXPORT void load(float *dst, float *src, unsigned count){hipMemcpy(dst, src, count*sizeof(float), hipMemcpyHostToDevice);};
EXPORT void numpy(float *dst, float *src, unsigned count){hipMemcpy(dst, src, count*sizeof(float), hipMemcpyDeviceToHost);};
EXPORT void fill(float *dst, unsigned count, float num){
    float *hostMem = (float*)malloc(sizeof(float)*count);
    for(int i=0; i<count; i++) hostMem[i] = num;
    hipMemcpy(dst, hostMem, count*sizeof(float), hipMemcpyHostToDevice);
}
EXPORT void freeMem(void *ptr){hipFree(ptr);};